#include "hip/hip_runtime.h"
#include <complex.h>
#include <hipfft/hipfft.h>
#include "phaser/backend/correlation/spatial-correlation-cuda.h"

#include <gflags/gflags.h>
#include <glog/logging.h>

DEFINE_double(phase_gpu_batch, 1, "");

namespace alignment {

SpatialCorrelationCuda::SpatialCorrelationCuda(const uint32_t voxels_per_dim)
    : n_voxels_total_(voxels_per_dim * voxels_per_dim * voxels_per_dim),
      n_voxels_per_dim_(voxels_per_dim) {
  // Allocate memory for the FFT and IFFT.
  const uint32_t data_size = sizeof(hipfftDoubleComplex) * n_voxels_total_;
  hipMalloc((void**)&F_, data_size);
  hipMalloc((void**)&G_, data_size);
  hipMalloc((void**)&C_, data_size);
  c_ = new double[n_voxels_total_];

  // Create the cuda plans for two FFTs and one IFFT.
  hipfftPlan3d(
      &f_plan_, voxels_per_dim, voxels_per_dim, voxels_per_dim, HIPFFT_D2Z);
  hipfftPlan3d(
      &c_plan_, voxels_per_dim, voxels_per_dim, voxels_per_dim, HIPFFT_Z2D);
}

SpatialCorrelationCuda::~SpatialCorrelationCuda() {
  hipFree(F_);
  hipFree(G_);
  hipFree(C_);
  hipfftDestroy(f_plan_);
  hipfftDestroy(c_plan_);
  delete[] c_;
}

// CUDA kernel for the spatial phase correlation.
// Simple complex multiplication version.
__global__ void correlation(
    hipfftDoubleComplex* F, hipfftDoubleComplex* G, hipfftDoubleComplex* C,
    uint32_t size) {
  // unique block index inside a 3D block grid
  const unsigned long long int blockId =
      blockIdx.x                             // 1D
      + blockIdx.y * gridDim.x               // 2D
      + gridDim.x * gridDim.y * blockIdx.z;  // 3D

  // global unique thread index, block dimension uses only x-coordinate
  const unsigned long long int idx = blockId * blockDim.x + threadIdx.x;

  C[idx].x = F[idx].x * G[idx].x - F[idx].y * (-G[idx].y);
  C[idx].y = F[idx].x * (-G[idx].y) + F[idx].y * G[idx].x;
}

double* SpatialCorrelationCuda::correlateSignals(
    double* const f, double* const g) {
  // Perform the two FFTs on the discretized signals.
  VLOG(1) << "Performing FFT on the first point cloud.";
  double* d_input;
  hipMalloc((void**)(&d_input), sizeof(double) * n_voxels_total_);
  hipMemcpy(
      d_input, f, sizeof(double) * n_voxels_total_, hipMemcpyHostToDevice);
  // R2C is for floats, D2Z for doubles.
  if (hipfftExecD2Z(f_plan_, d_input, F_) != HIPFFT_SUCCESS) {
    LOG(FATAL) << "Forward cufft transform failed.";
  }

  VLOG(1) << "Performing FFT on the second point cloud.";
  hipMemcpy(
      d_input, g, sizeof(double) * n_voxels_total_, hipMemcpyHostToDevice);
  hipfftExecD2Z(f_plan_, d_input, G_);

  // Correlate the signals in the frequency domain.
  const auto tile_width = 32;
  const auto n_threads = std::ceil((float)n_voxels_per_dim_ / tile_width);
  dim3 dimBlock(n_threads, n_threads, n_threads);
  dim3 dimGrid(tile_width, tile_width, 1);
  correlation<<<dimGrid, dimBlock>>>(F_, G_, C_, n_voxels_total_);

  // Perform the IFFT on the correlation tensor.
  VLOG(1) << "Performing IFFT on correlation.";
  hipfftExecZ2D(c_plan_, C_, d_input);
  hipMemcpy(
      c_, d_input, sizeof(double) * n_voxels_total_, hipMemcpyDeviceToHost);
  return c_;
}

}  // namespace alignment
