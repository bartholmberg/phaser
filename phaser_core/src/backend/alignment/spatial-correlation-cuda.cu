#include "hip/hip_runtime.h"
#include "phaser/backend/alignment/spatial-correlation-cuda.h"
#include <complex.h>
#include <hipfft/hipfft.h>

#include <gflags/gflags.h>
#include <glog/logging.h>

DEFINE_double(
    phase_gpu_batch, 1,
    "");

namespace alignment {

SpatialCorrelationCuda::SpatialCorrelationCuda(const uint32_t voxels_per_dim)
    : n_voxels_total_(voxels_per_dim * voxels_per_dim * voxels_per_dim),
      n_voxels_per_dim_(voxels_per_dim) {

  // Allocate memory for the FFT and IFFT.
  const uint32_t data_size = sizeof(hipfftComplex) * n_voxels_total_
    * FLAGS_phase_gpu_batch;
  hipMalloc((void**)&F_, data_size);
  hipMalloc((void**)&G_, data_size);
  hipMalloc((void**)&C_, data_size);
  c_ = new double[n_voxels_total_];

  // Create the cuda plans for two FFTs and one IFFT.
  hipfftPlan3d(&f_plan_, voxels_per_dim, voxels_per_dim, voxels_per_dim,
    HIPFFT_R2C);
  hipfftPlan3d(&c_plan_, voxels_per_dim, voxels_per_dim, voxels_per_dim,
    HIPFFT_C2R);
}

SpatialCorrelationCuda::~SpatialCorrelationCuda() {
  hipFree(F_);
  hipFree(G_);
  hipFree(C_);

  hipfftDestroy(f_plan_);
  hipfftDestroy(c_plan_);

  delete [] c_;
}

// CUDA kernel for the spatial phase correlation.
__global__ void correlation(hipfftDoubleComplex* F, hipfftDoubleComplex* G,
    uint32_t size) {
  const int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx >= size) return;

  F[idx].x = F[idx].x * G[idx].x - F[idx].y * (-G[idx].y);
  F[idx].y = F[idx].x * (-G[idx].y) + F[idx].y * G[idx].x;
}

void SpatialCorrelationCuda::correlateSignals(double* const f,
    double* const g) {
  // Perform the two FFTs on the discretized signals.
  VLOG(1) << "Performing FFT on the first point cloud.";
  double* d_input;
  hipMalloc((&d_input), sizeof(double)*n_voxels_total_);
  hipMemcpy(d_input, f, sizeof(double)*n_voxels_total_,
    hipMemcpyHostToDevice);
  // R2C is for floats, D2Z for doubles.
  hipfftExecD2Z(f_plan_, d_input, F_);

  VLOG(1) << "Performing FFT on the second point cloud.";
  hipMemcpy(d_input, g, sizeof(double)*n_voxels_total_,
    hipMemcpyHostToDevice);
  hipfftExecD2Z(f_plan_, d_input, G_);

  // Correlate the signals in the frequency domain.
  dim3 dimBlock(n_voxels_per_dim_, n_voxels_per_dim_, n_voxels_per_dim_);
  dim3 dimGrid(1,1,1);
  correlation<<<dimGrid, dimBlock>>>(F_, G_, n_voxels_total_);

  // Perform the IFFT on the correlation tensor.
  VLOG(1) << "Performing IFFT on correlation.";
  hipfftExecZ2D(c_plan_, F_, d_input);
  hipMemcpy(c_, d_input, sizeof(double)*n_voxels_total_, hipMemcpyDeviceToHost);
}

}  // namespace alignment
