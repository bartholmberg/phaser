#include "hip/hip_runtime.h"
#include "phaser/backend/alignment/phase-aligner-opt.h"
#include <complex.h>
#include <hipfft/hipfft.h>

#include <gflags/gflags.h>
#include <glog/logging.h>

DEFINE_double(
    phase_gpu_discretize_lower, -50,
    "Specifies the lower bound for the discretization.");
DEFINE_double(
    phase_gpu_discretize_upper, 50,
    "Specifies the upper bound for the discretization.");
DEFINE_double(
    phase_gpu_n_voxels, 200,
    "Specifies the number of voxels for the discretization.");
DEFINE_double(
    phase_gpu_batch, 1,
    "");

namespace alignment {

PhaseAlignerGpu::PhaseAlignerGpu()
    : n_voxels_(
          FLAGS_phase_gpu_n_voxels * FLAGS_phase_gpu_n_voxels
          * FLAGS_phase_gpu_n_voxels) {
  VLOG(1) << "Initializing phase alignment with " << FLAGS_phase_gpu_n_voxels
          << " voxels in [" << FLAGS_phase_gpu_discretize_lower << ", "
          << FLAGS_phase_gpu_discretize_upper << "].";

  // Allocate memory for the FFT and IFFT.
  uint32_t data_size = sizeof(hipfftComplex)*n_voxels_*FLAGS_phase_gpu_batch;
  hipMalloc((void**)&F_, data_size);
  hipMalloc((void**)&G_, data_size);
  hipMalloc((void**)&C_, data_size);
  c_ = new double[n_voxels_];

  // Allocate memory for the function signals in the time domain.
  //f_ = Eigen::VectorXd::Zero(n_voxels_);
  //g_ = Eigen::VectorXd::Zero(n_voxels_);

  // Create the cuda plans for two FFTs and one IFFT.
  hipfftPlan3d(&f_plan_, FLAGS_phase_gpu_n_voxels,
      FLAGS_phase_gpu_n_voxels, FLAGS_phase_gpu_n_voxels,
      HIPFFT_R2C);
   hipfftPlan3d(&c_plan_, FLAGS_phase_gpu_n_voxels,
      FLAGS_phase_gpu_n_voxels, FLAGS_phase_gpu_n_voxels,
      HIPFFT_C2R);
}

PhaseAlignerGpu::~PhaseAlignerGpu() {
  hipFree(F_);
  hipFree(G_);
  hipFree(C_);

  hipfftDestroy(f_plan_);
  hipfftDestroy(c_plan_);

  //fftw_cleanup();
  delete [] c_;
}

__global__ void correlation(hipfftComplex* F, hipfftComplex* G, uint32_t size) {
  const int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx >= size) return;

  F[idx].x = F[idx].x * G[idx].x - F[idx].y * (-G[idx].y);
  F[idx].x = F[idx].x * (-G[idx].y) + F[idx].y * G[idx].x;
}

  /*
void PhaseAlignerGpu::alignRegistered(
    const model::PointCloud& cloud_prev,
    const std::vector<model::FunctionValue>&,
    const model::PointCloud& cloud_reg,
    const std::vector<model::FunctionValue>&, common::Vector_t* xyz) {
  CHECK(xyz);
  auto start = std::chrono::high_resolution_clock::now();
  discretizePointcloud(cloud_prev, &f_, &hist_);
  discretizePointcloud(cloud_reg, &g_, &hist_);

  // Perform the two FFTs on the discretized signals.
  VLOG(1) << "Performing FFT on the first point cloud.";
  double* d_input;
  hipMalloc((&d_input), sizeof(double)*n_voxels_);
  hipMemcpy(d_input, f_.data(), sizeof(double)*n_voxels_, hipMemcpyHostToDevice);
  hipfftExecR2C(f_plan_, d_input, F_);
  VLOG(1) << "Performing FFT on the second point cloud.";
  hipMemcpy(d_input, g_.data(), sizeof(double)*n_voxels_, hipMemcpyHostToDevice);
  hipfftExecR2C(f_plan_, d_input, G_);

  // Correlate the signals in the frequency domain.
  const uint32_t dim = FLAGS_phase_gpu_n_voxels;
  dim3 dimBlock(dim, dim, dim);
  dim3 dimGrid(1,1,1);
  correlation<<<dimGrid, dimBlock>>>(F_, G_, n_voxels_);

  // Perform the IFFT on the correlation tensor.
  VLOG(1) << "Performing IFFT on correlation.";
  double* d_input;
  hipfftExecR2C(c_plan_, F_, d_input);
  hipMemcpy(c_, d_input, sizeof(double)*n_voxels_, hipMemcpyDeviceToHost);

  // Find the index that maximizes the correlation.
  auto end = std::chrono::high_resolution_clock::now();
  double duration =
      std::chrono::duration_cast<std::chrono::milliseconds>(end - start)
          .count();
  std::cout << "Translation alignment times: " << duration << " \n";
}
  */

std::vector<double> PhaseAlignerGpu::getCorrelation() const {
  return std::vector<double>(c_, c_+n_voxels_);
}

}  // namespace alignment
